
#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <iostream>
#include "DataSet.h"
#include "NNetwork.h"

int main()
{
	srand(time(NULL));
	std::cout << "hello\n";
	
	DataSet d;

	NNetwork net;
	net.addLayer(new Layer("hiddenLayer1", 28 * 28, 32, ActivationTypes::ReLU));

	CustomCudaArray Z = net.forward(d.training_data.front().image);

	int a = 1;
	
	return 0;
}
