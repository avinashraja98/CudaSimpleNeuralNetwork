
#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <iostream>
#include "DataSet.h"
#include "NNetwork.h"

int main()
{
	srand(time(NULL));
	std::cout << "hello\n";
	
	DataSet d;

	NNetwork net;
	net.addLayer(new Layer("hiddenLayer1", 28 * 28, 32, ActivationTypes::ReLU));
	
	return 0;
}
