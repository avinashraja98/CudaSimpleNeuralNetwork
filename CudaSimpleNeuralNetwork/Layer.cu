#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "Layer.h"
#include "ReLUActivation.h"

#include <assert.h>
#include <random>

__global__ void forwardKernel(float* W, float* A, float* Z, float* b,
	int W_x_dim, int W_y_dim,
	int A_x_dim, int A_y_dim) {

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int Z_x_dim = W_x_dim;
	int Z_y_dim = A_y_dim;

	float Z_value = 0;

	if (row < Z_x_dim && col < Z_y_dim) {
		/*
		for (int i = 0; i < W_y_dim; i++) {
			Z_value += W[row * W_y_dim + i] * A[i * A_y_dim + col];
		}
		Z[row * Z_y_dim + col] = Z_value + b[row];
		*/
		Z[0] = 1.0f;
		int a = 1;
	}
}

void Layer::fillWithZeroes(CustomCudaArray & inp)
{
	size_t inpSize = inp.getSize();
	for (int i = 0; i < inpSize; i++) {
		inp[i] = (float)0;
	}
}

void Layer::fillWithRandom(CustomCudaArray & inp)
{
	std::default_random_engine generator;
	std::normal_distribution<float> normal_distribution(0.0, 1.0);
	
	size_t inpSize = inp.getSize();

	for (int i = 0; i < inpSize; i++) {
		inp[i] = normal_distribution(generator) * random_init_threshold;
	}
}

Layer::Layer(std::string name, size_t numInputs, size_t numUnits, ActivationTypes actType) : W(numUnits, numInputs), b(numUnits)
{
	this->name = name;
	this->numUnits = numUnits;
	this->numInputs = numInputs;

	fillWithZeroes(b);
	fillWithRandom(W);

	switch (actType)
	{
	case ActivationTypes::ReLU:
		activation = new ReLUActivation();
		break;
	default:
		activation = new ReLUActivation();
		break;
	}
}

Layer::~Layer()
{
	delete activation;
}

CustomCudaArray & Layer::forward(CustomCudaArray & A)
{
	// Z = W.A + b
	// For matrix multiplication
	assert(W.getY() == A.getX());

	Z[0] = 0.0;
	this->A = A;

	Z.resizeAndReset(W.getX(), A.getY());

	dim3 block_size(16, 16);
	
	dim3 num_of_blocks(((unsigned int)W.getX() + block_size.x - 1) / block_size.x,
		((unsigned int)A.getY() + block_size.y - 1) / block_size.y);

	forwardKernel<<<num_of_blocks, block_size >>>(W.dataPtr.get(),
		A.dataPtr.get(),
		Z.dataPtr.get(),
		b.dataPtr.get(),
		(int)W.getX(), (int)W.getY(),
		(int)A.getX(), (int)A.getY());

	hipDeviceSynchronize();

	return Z;
}
